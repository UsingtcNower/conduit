#include "cudautil.hpp"

#include <hip/hip_runtime.h>

bool CudaUtil::hasCuda() {
  int deviceCount, device;
  int gpuDeviceCount = 0;
  struct hipDeviceProp_t properties;
  hipError_t cudaResultCode = hipGetDeviceCount(&deviceCount);
  if (cudaResultCode != hipSuccess) {
    deviceCount = 0;
  }
  // Machines with no GPUs can still report one emulation device
  for (device = 0; device < deviceCount; device++) {
    hipGetDeviceProperties(&properties, device);
    if (properties.major != 9999) {
      // 9999 means emulation only
      gpuDeviceCount++;
    }
  }
  return gpuDeviceCount > 0;
}
