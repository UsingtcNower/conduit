#include "hip/hip_runtime.h"
#include "buildtest.h"

#include <iostream>

namespace BuildTest {
  __global__ void computeIndex(int* arr, int N) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < N) {
      arr[i] = i;
    }
  }

  int runBuildTest() {
    std::cout << "Running the tests..." << std::endl;

    const int N = 1 << 12;
    const int THREADS_PER_BLK = 64;

    int* arr;
    hipMalloc((void**)&arr, N * sizeof(int));

    computeIndex<<<N/THREADS_PER_BLK, THREADS_PER_BLK>>>(arr, N);

    int localArr[N];

    hipMemcpy(localArr, arr, N * sizeof(int), hipMemcpyDeviceToHost);

    int numErrors = 0;
    for (int i = 0; i < N; i++) {
      if (i != localArr[i]) {
        numErrors++;
      }
    }

    if (numErrors == 0) {
      std::cout << "+++ CUDA build test passed! +++" << std::endl;
    } else {
      std::cerr << "--- FAILED CUDA BUILD TEST ---" << std::endl;
    }

    return numErrors;
  }
}
